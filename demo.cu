#include "hip/hip_runtime.h"
#include "bind.hpp"
#include <iostream>

__host__ __device__
void demo()
{
  using namespace thrust::placeholders;
  using namespace thrust::experimental;

  auto return_13 = bind([]
  {
    return 13;
  });

  printf("return_13: %d\n", return_13());

  auto add_1 = bind([](int x)
  {
    return x + 1;
  },
  _1
  );

  printf("add_1(0): %d\n", add_1(0));

  auto sum = [](int x, int y)
  {
    return x + y;
  };

  auto sum_two_and_three = bind(sum, 2, 3);

  printf("sum_two_and_three: %d\n", sum_two_and_three());

  auto bound_sum = bind(sum, _1, _2);

  printf("bound_sum(2,3): %d\n", bound_sum(2,3));
}

__global__ void device_demo()
{
  printf("Demo on the device\n");
  demo();
}

int main()
{
  printf("Demo on the host\n");
  demo();

  device_demo<<<1,1>>>();
  hipDeviceSynchronize();

  return 0;
}

